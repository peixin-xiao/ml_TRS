#include "hip/hip_runtime.h"
﻿//#pragma comment( linker, "/subsystem:windows /entry:mainCRTStartup" )
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>


#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include <algorithm>
#include <tuple>
#include <random>
#include <iostream>
#include <fstream>
#include <cstdlib> 
#include <ctime>  

#include "./model_gen.h"

#define M 200//amount of a batch of photons
#define AM 5000
#define N int(100) // amount of vertex of model geometry

#define K int(1e5)
#define X 2 //kinds of participate medium
#define DELTA_T     1e-14  //1e-14
#define	PI          3.1415926
#define	LIGHTSPEED	2.997925E8 /* in vacuo speed of light [m/s] */
#define ALIVE       1   		/* if photon not yet terminated */
#define DEAD        0    		/* if photon is to be terminated */
#define THRESHOLD   1e-4		/* used in roulette */
#define CHANCE      0.1  		/* used in roulette */
#define ONE_MINUS_COSZERO 1.0E-12
/* If 1-cos(theta) <= ONE_MINUS_COSZERO, fabs(theta) <= 1e-6 rad. */
/* If 1+cos(theta) <= ONE_MINUS_COSZERO, fabs(PI-theta) <= 1e-6 rad. */
#define SIGN(x)           ((x)>=0 ? 1:-1)

//struct Tissue {
//    int    index;
//    char   name[10];
//    float miu_a;
//    float miu_s;
//    float g;
//    float index_of_refraction;
//};

__device__ thrust::tuple<float(*)[N][2]> bubbleSort(float(*ar)[N][2], int FLAG, int idx) {

    for (int i = 0; i < FLAG - 1; i++) {
        for (int j = 0; j < FLAG - 1 - i; j++) {
            if (ar[idx][j][0] > ar[idx][j + 1][0]) {
                float temp0 = ar[idx][j + 1][0];
                ar[idx][j + 1][0] = ar[idx][j][0];
                ar[idx][j][0] = temp0;
                float temp1 = ar[idx][j + 1][1];
                ar[idx][j + 1][1] = ar[idx][j][1];
                ar[idx][j][1] = temp1;
            }
        }
        //printf("this is %d:", i + 1);

    }

    return thrust::make_tuple(ar);
}

__device__ thrust::tuple<float, float, float> refra(float(*face1)[3], float(*face2)[3]
    , float(*face3)[3], float refra1, float refra2, float ux, float uy, float uz, float weight,
    float(*d1)[3], float(*d2)[3], float(*di)[3], float(*n1)[3], float(*n2)[3],
    float(*vrot)[3], float(*temp0)[3], float(*temp2)[3], int idx) {

    /*float* d1, * d2, * di, * n1, * n2, * vrot, * temp0, * temp2;
    hipMalloc((void**)&d1, 3 * sizeof(float));
    hipMalloc((void**)&d2, 3 * sizeof(float));
    hipMalloc((void**)&di, 3 * sizeof(float));
    hipMalloc((void**)&n1, 3 * sizeof(float));
    hipMalloc((void**)&n2, 3 * sizeof(float));
    hipMalloc((void**)&vrot, 3 * sizeof(float));
    hipMalloc((void**)&temp0, 3 * sizeof(float));
    hipMalloc((void**)&temp2, 3 * sizeof(float));*/
    int i;
    float tempn1, theta, theta1, theta2, temp1;

    for (i = 0;i < 3;i++) {
        d1[idx][i] = face1[idx][i] - face2[idx][i];
        d2[idx][i] = face2[idx][i] - face3[idx][i];
    }

    di[idx][0] = ux;/*light incident direction*/
    di[idx][1] = uy;
    di[idx][2] = uz;

    n1[idx][0] = d1[idx][1] * d2[idx][2] - d1[idx][2] * d2[idx][1];
    n1[idx][1] = d1[idx][2] * d2[idx][0] - d1[idx][0] * d2[idx][2];
    n1[idx][2] = d1[idx][0] * d2[idx][1] - d1[idx][1] * d2[idx][0];//normal line of the face : n1 = d1 × d2
    //----------------------------------------------------
    tempn1 = pow(n1[idx][0], 2) + pow(n1[idx][1], 2)
        + pow(n1[idx][2], 2);
    tempn1 = pow(tempn1, 0.5);
    n1[idx][0] = n1[idx][0] / tempn1;
    n1[idx][1] = n1[idx][1] / tempn1;
    n1[idx][2] = n1[idx][2] / tempn1;//make the norm of vector n1 to 1
    //-----------------------------------------------------
    if (n1[idx][0] * di[idx][0] + n1[idx][1] * di[idx][1] + n1[idx][2] * di[idx][2] < 0)
    {
        for (i = 0;i < 3;i++) n1[idx][i] = -n1[idx][i]; /*inverse normal to make sure
                                                        normal of the face is in the
                                                        same direction as light direction*/
    }

    n2[idx][0] = n1[idx][1] * di[idx][2] - n1[idx][2] * di[idx][1];
    n2[idx][1] = n1[idx][2] * di[idx][0] - n1[idx][0] * di[idx][2];
    n2[idx][2] = n1[idx][0] * di[idx][1] - n1[idx][1] * di[idx][0];/*normal line between the light direction and
                                                   the normal line of the face : n2 = n1 × light
                                                   direction*/
                                                   //----------------------------------------------------
    tempn1 = pow(n2[idx][0], 2) + pow(n2[idx][1], 2)
        + pow(n2[idx][2], 2);
    tempn1 = pow(tempn1, 0.5);
    n2[idx][0] = n2[idx][0] / tempn1;
    n2[idx][1] = n2[idx][1] / tempn1;
    n2[idx][2] = n2[idx][2] / tempn1;//make the norm of vector n1 to 1
    //-----------------------------------------------------
    theta1 = acos(n1[idx][0] * di[idx][0] + n1[idx][1] * di[idx][1] + n1[idx][2] * di[idx][2]);
    theta2 = asin(refra1 * sin(theta1) / refra2);

    if (theta2 - theta1 > 0) theta = theta2 - theta1;
    else theta = 2 * PI + theta2 - theta1;
    /* vector di rotate by angle theta about vector n2*/
    temp0[idx][0] = n2[idx][1] * di[idx][2] - n2[idx][2] * di[idx][1];
    temp0[idx][1] = n2[idx][2] * di[idx][0] - n2[idx][0] * di[idx][2];
    temp0[idx][2] = n2[idx][0] * di[idx][1] - n2[idx][1] * di[idx][0];/* temp = n2 × di*/

    temp1 = n2[idx][0] * di[idx][0] + n2[idx][1] * di[idx][1] + n2[idx][2] * di[idx][2];
    temp2[idx][0] = n2[idx][0] * temp1;
    temp2[idx][1] = n2[idx][1] * temp1;
    temp2[idx][2] = n2[idx][2] * temp1;

    for (i = 0;i < 3;i++) {
        vrot[idx][i] = di[idx][i] * cos(theta) + temp0[idx][i] * sin(theta)
            + temp2[idx][i] * (1 - cos(theta));
    }

    return thrust::make_tuple(vrot[idx][0], vrot[idx][1], vrot[idx][2]);
}

std::tuple<int, float, short int*, int, int(*)[3]> read_model(std::string directory) {
    int i, j, k;
    int flag, FLAG;
    int(*p)[3];
    static int p1[N][3];
    int Nbins;
    float length_voxel;
    short int* T;
    std::ifstream fin(directory, std::ios::binary);
    fin.read((char*)&Nbins, sizeof(int));
    fin.read((char*)&length_voxel, sizeof(float));
    T = (short int*)malloc(1 * pow(Nbins, 3) * sizeof(short int));
    for (i = 0; i < Nbins;i++) {
        for (j = 0;j < Nbins;j++) {
            for (k = 0;k < Nbins;k++) {

                fin.read((char*)&(*(T + i * Nbins * Nbins +
                    j * Nbins + k)), sizeof(short int));

            }
        }
    }
    fin.read((char*)&(FLAG), sizeof(int));
    for (flag = 0;flag < FLAG;flag++) {
        fin.read((char*)&(p1[flag][0]), sizeof(int));
        fin.read((char*)&(p1[flag][1]), sizeof(int));
        fin.read((char*)&(p1[flag][2]), sizeof(int));
    }
    fin.close();
    p = p1;
    return std::make_tuple(Nbins, length_voxel, T, FLAG, p);
    std::cout << "model written\n";
}


unsigned int Rand0(unsigned int randx)
{
    randx = randx * 1103515245 + 12345;
    return randx & 2147483647;
}

unsigned int Rand01(unsigned int randx)
{
    randx = randx * 1664525 + 1013904223;
    return randx & 4294967296;
}

__device__
unsigned int Rand1(unsigned int randx)
{
    randx = randx * 1103515245 + 12345;
    return randx & 2147483647;
}


__device__ thrust::tuple<float, bool, int, float, int, int, int, float(*)[K][5]>distance_to_interface(float x,
    float y, float z, float ux, float uy, float uz, float w,
    short int* T, float delta_t, int Nbins, float s,
    float length_voxel, Tissue a[X], int time, float(*co)[K][5], bool reset, int idx) {

    float xr = x, yr = y, zr = z;
    float s1 = 0;
    float weight = w;
    //static float co[K][5] = { 0 };
    //float(*p)[5];
    int flag = 1;
    int i = 0, j = 0;
    int if_refraction = 0;
    int if_scatter = 0;
    //static int time = 0;
    bool if_beyond;
    short int type0 = T[int(x / length_voxel) * Nbins * Nbins
        + int(y / length_voxel) * Nbins + int(z / length_voxel)];
    if_beyond = 0;
    float refraction = a[type0].index_of_refraction;
    float mua = a[type0].miu_a;
    float step = LIGHTSPEED * delta_t / refraction;
    float length_of_side = Nbins * length_voxel;

    if (reset) {
        time = 0;
        memset(co, 0, sizeof(co));
    }

    do {

        if (((xr / length_voxel) - 1) < 0 || ((yr / length_voxel) - 1) < 0
            || ((zr / length_voxel) - 1) < 0 ||
            ((xr / length_voxel) - 1) > Nbins - 1 ||
            ((yr / length_voxel) - 1) > Nbins - 1 ||
            ((zr / length_voxel) - 1) > Nbins - 1)
        {
            flag = 0;
            if_beyond = 1;
            //std::cout << "out of cube\n";
        }
        else
        {

            if (T[int((xr + step * ux) / length_voxel) * Nbins * Nbins +
                int((yr + step * uy) / length_voxel) * Nbins +
                int((zr + step * uz) / length_voxel)] == type0)
            {
                s1 += step;
                xr = xr + step * ux;
                yr = yr + step * uy;
                zr = zr + step * uz;
                weight = weight * (exp(-mua * step));
                if (i % 100 == 0) {
                    co[idx][j][0] = xr;
                    co[idx][j][1] = yr;
                    co[idx][j][2] = zr;
                    co[idx][j][3] = weight;
                    co[idx][j][4] = delta_t * time;
                    j++;
                }
                time++;
                i++;
            }
            else {
                //s1 += length_voxel;
                int la = 1;
                xr = xr + la * step * ux;
                yr = yr + la * step * uy;
                zr = zr + la * step * uz;
                if (i % 100 == 0) {
                    co[idx][j][0] = xr;
                    co[idx][j][1] = yr;
                    co[idx][j][2] = zr;
                    co[idx][j][3] = weight;
                    co[idx][j][4] = delta_t * time;
                    j++;
                }
                time++;
                i++;
                flag = 0;
                if_refraction = 1;
                //std::cout << "reach the interface\n";

            }
        }
        if (s1 >= s) {
            flag = 0;
            if_scatter = 1;
            //std::cout << "out of max length\n";
        }
        //std::cout << "flag = " << flag << "\n";
    } while (flag);
    //p = co;
    //std::cout << "i = " << i << "\n";
    return thrust::make_tuple(s1, if_beyond, j,
        weight, if_refraction, if_scatter, time, co);
}

__device__ thrust::tuple<float(*)[K][5], int> record_path(float(*c0)[K][5],
    int k, float(*path)[K][5], int k0, int reset, int idx) {
    //auto static  path = new float[K][5];

    //static int k0 = 0;
    int i, j;
    if (reset == 0) {

        for (i = 0;i < 5;i++) {
            //path[i].resize((k0 + k) * sizeof(float));
            //if(k0 ==0) path[0][i] = k;

            for (j = 0;j < k;j++) {
                path[idx][k0 + j + 1][i] = c0[idx][j][i];
            }
            //c00[i] = c0[j][i];
        }

    }
    else if (reset == 1) {

        memset(path, 0, sizeof(path));
        k0 = 0;

    }
    k0 = k0 + k;
    for (i = 0;i < 5;i++) path[idx][0][i] = k0;
    return thrust::make_tuple(path, k0);

    //k0 = k0 + k;

}



__device__ thrust::tuple< float(*)[K][5] >  montecarlo
(short int* T, int Nbins, float length_voxel, Tissue a[X], int FLAG,
    int(*vertex)[3], float(*path2_gpu)[K][5], float(*c0)[K][5], int rndseed, int idx
    , float(*diss)[N][2], float(*face1)[3], float(*face2)[3],
    float(*face3)[3], float(*d1)[3], float(*d2)[3], float(*di)[3], float(*n1)[3], float(*n2)[3],
    float(*vrot)[3], float(*temp0)[3], float(*temp2)[3]) {

    /* propagation parameters */
    float	x, y, z;        /* photon position */
    float	ux, uy, uz;     /* photon trajectory as cosines */
    double  uxx, uyy, uzz;	/* temporary values used during spin */
    double	s;              /* step sizes. s = -log(rnd)/mus [cm] */
    double  sleft = 0;          /* dimensionless */
    double	costheta;       /* cos(theta) */
    double  sintheta;       /* sin(theta) */
    double	cospsi;         /* cos(psi) */
    double  sinpsi;         /* sin(psi) */
    double	psi;            /* azimuthal angle */
    long	i_photon;       /* current photon */
    double	w;              /* photon weight */
    double	absorb;         /* weighted deposited in a step due to absorption */
    short   photon_status;  /* flag = alive=1 or dead=0 */

    /* other variables */
    double	mua;            /* absorption coefficient [cm^-1] */
    double	mus;            /* scattering coefficient [cm^-1] */
    double	g;              /* anisotropy [-] */
    double  refraction;     /* index of refraction*/
    double	nphotons;       /* number of photons in simulation */

    /* dummy variables */
    float  rnd;            /* assigned random value 0-1 */

    int i_voxel;
    bool flag1 = 0;
    double 	temp;           /* dummy variable */

    /* mcxyz bin variables */
    //float   length_voxel = 0.002;/*bins size, unit : [m] */
    //float   step = 0.5;
    //float   stepsize = length_voxel * step;
    double   delta_t = DELTA_T;

    float s1 = 0;
    float s3; //distance from initial point to the current position
    //int s_block;
    int if_refraction = 0;
    int if_scatter = 0;

    //float c0[K][5];
    //auto c0 = new float[K][5];
    //hipMemset(c0, 0, K * 5 * sizeof(float));
    //extern __shared__ float c0[K][5];


    int k0 = 0;

    int if_beyond, k;
    int i, j;
    int rndflag = 0;
    //auto rand = new float[COUNT];
    int n_rand = rndseed;

    //float (*dis)[2];
    //hipMalloc((void**) & dis, N * 2 * sizeof(float));
    //float(*dis2)[2]; // all surface point before and after bubble sort
    ////float face1[3], face2[3], face3[3]; // coordinates of the 3 nearest point
    //float* face1;
    //float* face2;
    //float* face3;
    //hipMalloc((void**)&face1, 3 * sizeof(float));
    //hipMalloc((void**)&face2, 3 * sizeof(float));
    //hipMalloc((void**)&face3, 3 * sizeof(float));
    int pos; //position of the minimum 3 point near the light penetration point
    float refra0, refra1; //refraction index before and after refraction

    //thrust::tie(n_rand,rnd,rand) = randnum(n_rand,rand,0)

    nphotons = M - 1; // will be updated to achieve desired run time, time_min.
    i_photon = -1;


    i_photon += 1;				/* increment photon count */
    w = 1.0;                    /* set photon weight to one */
    photon_status = ALIVE;      /* launch an alive photon */

    x = 0.01 * Nbins * length_voxel;
    y = 0.5 * Nbins * length_voxel;
    z = 0.5 * Nbins * length_voxel;
    ux = 1;
    uy = 0;
    uz = 0;
    sleft = 0;
    //std::cout << "\n\n\nphoton : " << i_photon << " launched\n";
    int loop = 0;
    int time = 0;

    /*distance_to_interface(x, y, z, ux, uy, uz, w, T,
        delta_t, Nbins, 0, length_voxel, a,time,c0, 1);*/


    do {
        loop++;


        if (sleft == 0 || flag1 == 1) {

            //thrust::tie(n_rand, rnd, rand) = randnum(n_rand, rand, 0);
            n_rand = Rand1(n_rand);
            rnd = float(n_rand) / 2147483647;

            sleft = -log(rnd);

            flag1 = 0;
        }
        else {
            i_voxel = int(x / length_voxel) * Nbins * Nbins
                + int(y / length_voxel) * Nbins + int(z / length_voxel);
            mua = a[T[i_voxel]].miu_a;
            mus = a[T[i_voxel]].miu_s;
            g = a[T[i_voxel]].g;
            refraction = a[T[i_voxel]].index_of_refraction;

            s = sleft / (mus * (1 - g) + mua);  //[m]
            //s = sleft / (mus );  //[m]

            thrust::tie(s1, if_beyond, k, w,
                if_refraction, if_scatter, time, c0) =
                distance_to_interface(x, y, z, ux, uy, uz, w, T,
                    delta_t, Nbins, s, length_voxel, a, time, c0, 0, idx);
            //std::cout << "\n\nw = " << w << "\n\n";

            /*i_voxel = int(round((x / length_voxel) * Nbins * Nbins +
                (y / length_voxel) * Nbins + (z / length_voxel)));*/
            refra0 = a[T[i_voxel]].index_of_refraction;
            thrust::tie(path2_gpu, k0) = record_path(c0, k, path2_gpu, k0, 0, idx);

            /*std::cout << "c0.time : \n" ;
            for (i = 0;i < 10;i++) {
                std::cout << c0[i][4]<<"\n";
            }
            std::cout << "\n\n";*/
            //path1[i_photon] = path0;


            if (if_beyond == 1) { photon_status = DEAD; }
            if (s1 >= s) { s3 = s; flag1 = 1; }
            else s3 = s1;

            //std::cout << "s3 = " << s3 << " " << "s1 = " << s1 << "\n";

            k--;
            //if (k > 0) k--;
            /*int la = 1;
            x = c0[k][0] + la * length_voxel * ux;
            y = c0[k][1] + la * length_voxel * uy;
            z = c0[k][2] + la * length_voxel * uz;*/
            x = c0[idx][k][0];
            y = c0[idx][k][1];
            z = c0[idx][k][2];

            /*i_voxel = int(floor(((x / length_voxel)-1) * Nbins * Nbins +
                ((y / length_voxel)-1) * Nbins + ((z / length_voxel)-1)));*/
            i_voxel = int(x / length_voxel) * Nbins * Nbins
                + int(y / length_voxel) * Nbins + int(z / length_voxel);

            //std::cout << "xyz=" << x << y << z << "\n";
            if (i_voxel > pow(Nbins, 3) - 1) photon_status = DEAD;
            if (x <= 0 || y <= 0 || z <= 0 || x >=
                Nbins * length_voxel || y >= Nbins * length_voxel ||
                z >= Nbins * length_voxel) {
                photon_status = DEAD;
            }
            if (i_voxel > sizeof(T) / sizeof(T[0])) i_voxel = sizeof(T) / sizeof(T[0]);
            refra1 = a[T[i_voxel]].index_of_refraction;

            //sleft = sleft - s3 * mus;
            sleft = sleft - s3 * (mus * (1 - g) + mua);

        }

        if (if_refraction == 1) {
            for (i = 0;i < FLAG;i++) {
                diss[idx][i][0] = pow((pow(x - length_voxel * vertex[i][0], 2)
                    + pow(y - length_voxel * vertex[i][1], 2)
                    + pow(z - length_voxel * vertex[i][2], 2)), 0.5);
                diss[idx][i][1] = i;
            }

            thrust::tie(diss) = bubbleSort(diss, FLAG, idx);
            pos = diss[idx][N - FLAG][1];
            //std::cout << "pos = " << pos << std::endl;
            for (i = 0;i < 3;i++) {
                face1[idx][i] = length_voxel * vertex[pos][i];
            }
            pos = diss[idx][N - FLAG + 1][1];
            for (i = 0;i < 3;i++) {
                face2[idx][i] = length_voxel * vertex[pos][i];
            }
            pos = diss[idx][N - FLAG + 2][1];
            for (i = 0;i < 3;i++) {
                face3[idx][i] = length_voxel * vertex[pos][i];
            }
            thrust::make_tuple(ux, uy, uz) =
                refra(face1, face2, face3, refra0, refra1, ux, uy, uz, w,
                    d1, d2, di, n1, n2, vrot, temp0, temp2, idx);

        }

        if (if_scatter == 1) {
            /* sample for costheta */
            n_rand = Rand1(n_rand);
            rnd = float(n_rand) / 2147483647;
            if (g == 0.0)
                costheta = 2.0 * rnd - 1.0;
            else {
                temp = (1.0 - g * g) / (1.0 - g + 2 * g * rnd);
                costheta = (1.0 + g * g - temp * temp) / (2.0 * g);
            }
            sintheta = sqrt(1.0 - costheta * costheta); /* sqrt() is faster than sin(). */

            /* sample psi. */
            n_rand = Rand1(n_rand);
            rnd = float(n_rand) / 2147483647;
            psi = 2.0 * PI * rnd;
            //psi = 2.0 * PI * RandomNum;
            cospsi = cos(psi);
            if (psi < PI)
                sinpsi = sqrt(1.0 - cospsi * cospsi);     /* sqrt() is faster than sin(). */
            else
                sinpsi = -sqrt(1.0 - cospsi * cospsi);

            /* new trajectory. */
            if (1 - fabs(uz) <= ONE_MINUS_COSZERO) {      /* close to perpendicular. */
                uxx = sintheta * cospsi;
                uyy = sintheta * sinpsi;
                uzz = costheta * SIGN(uz);   /* sign() is faster than division. */
            }
            else {					/* usually use this option */
                temp = sqrt(1.0 - uz * uz);
                uxx = sintheta * (ux * uz * cospsi - uy * sinpsi) / temp + ux * costheta;
                uyy = sintheta * (uy * uz * cospsi + ux * sinpsi) / temp + uy * costheta;
                uzz = -sintheta * cospsi * temp + uz * costheta;
            }

            /* update trajectory */
            ux = uxx;
            uy = uyy;
            uz = uzz;

            //std::cout << "new direction = " << ux << "," << uy << "," << uz << "\n";

        }
        /**** check roulette
             if photon weight below threshold, then terminate photon using roulette technique.
             photon has chance probability of having its weight increased by factor of 1/chance,
             and 1-chance probability of terminating.
             *****/
        if (w < THRESHOLD) {
            n_rand = Rand1(n_rand);
            rnd = float(n_rand) / 2147483647;
            if (rnd <= CHANCE)
                w /= CHANCE;
            else
            {
                photon_status = DEAD;
                //std::cout << "i_photon = " << i_photon << "\n";
                //std::cout << "coordinate = " << x << "," << y << "," << z << "\n";

            }
        }






    } while (photon_status == ALIVE);  /* end step_check_hop_spin */
    /* if alive, continue propagating */
    /* if photon dead, then launch new photon. */

    /*std::cout << "path0.time : \n" ;
            for (i = 0;i < 100;i++) {
                std::cout << path0[i][4]<<"\n";
            }*/
            /*for (i = 0;i < K;i++) {
                for (j = 0;j < 5;j++) {
                    path1[i_photon][i][j] = path0[i][j];
                }
            }*/

            //record_path(c0, k, path0,k0,1);




    return thrust::make_tuple(path2_gpu);

}

__global__ void simulate(short int* T, int Nbins, float length_voxel, Tissue a[X], int FLAG,
    int(*vertex)[3], float(*path2_gpu)[K][5], float(*c0)[K][5],
    int* rndseed_gpu, float(*diss)[N][2], float(*face1)[3], float(*face2)[3],
    float(*face3)[3], float(*d1)[3], float(*d2)[3], float(*di)[3], float(*n1)[3], float(*n2)[3],
    float(*vrot)[3], float(*temp0)[3], float(*temp2)[3]) {

    int i = threadIdx.x;
    //  int i = 0;
    int j, k;
    if (i < M) {

        thrust::tie(path2_gpu) = montecarlo(T, Nbins, length_voxel, a, FLAG,
            vertex, path2_gpu, c0, rndseed_gpu[i], i, diss, face1, face2, face3,
            d1, d2, di, n1, n2, vrot, temp0, temp2);




    }
    //return thrust::make_tuple(path2_gpu);
}

void write_path(std::string directory) {
    std::ofstream fout(directory, std::ios::binary);
    int i, j, k;
    int I, J, L;
    I = AM;
    //I = 2;
    J = 5;
    L = 50;
    fout.write((char*)&(I), sizeof(int));
    fout.write((char*)&(J), sizeof(int));
    fout.write((char*)&(L), sizeof(int));
    //for (i = 0;i < M;i++) {
    //    for (j = 0;j < J;j++) {
    //        for (k = 0;k < path[i][j].size();k++) {
    //            fout.write((char*)&path[i][j][k], sizeof(float));
    //            //std::cout << path[i][j][k] << "\n";
    //        }
    //        fout.write("end", 4);
    //    }
    //}

    fout.close();
}

void write_path3(std::string directory) {
    std::ofstream fout(directory, std::ios_base::binary |
        std::ios_base::app | std::ios_base::out);

    int I;
    I = 800;
    fout.seekp(0, SEEK_SET);
    fout.write((char*)&(I), sizeof(int));

    fout.close();
}

void write_path2(std::vector< std::vector<float> >(*path), std::string directory) {
    using namespace std;
    //std::fstream fout(directory, std::ios::binary| ios::out | ios::in);
    //ofstream fout(directory, ios::out | ios::binary);
    //ofstream fout(directory, ios_base::app);
    std::ofstream fout(directory, std::ios_base::binary | ios_base::app | ios_base::out);
    //std::ofstream fout(directory, ios_base::app | ios_base::out);

    int i, j, k;
    int I, J, L;
    I = M;
    J = 5;
    L = 50;
    //std::cout << "size = " << path[0][0][0];
    for (i = 0;i < M;i++) {
        for (j = 0;j < J;j++) {
            for (k = 0;k < path[i][j].size();k++) {
                fout.write((char*)&path[i][j][k], sizeof(float));
                //std::cout << path[i][j][k] << "\n";
            }
            fout.write("end", 4);
        }
    }

    fout.close();

}

std::vector< std::vector<float> >* array2vector(float(*path1)[K][5],
    bool reset) {
    int i, j, l;
    int k;
    static std::vector< std::vector<float> > path[M];
    if (reset == 0) {
        for (i = 0;i < M;i++) {
            for (j = 0;j < 5;j++) {
                k = path1[i][0][j];
                path[i].resize(5 * sizeof(float));
                for (l = 1;l < k;l++) {
                    path[i][j].push_back(path1[i][l][j]);
                }
            }

        }
    }
    else {
        for (i = 0;i < M;i++) path[i].clear();
    }

    //std::cout << "size = " << path[0][4].size();
    //std::cout << "010101\n";
    return path;
}

//std::tuple<float,float,float> read_para(std::string dic) {
//    float a[4];
//    std::ifstream fin(dic, std::ios::binary);
//
//    fin.read((char*)&a[0], sizeof(float));
//    fin.read((char*)&a[1], sizeof(float));
//    fin.read((char*)&a[2], sizeof(float));
//    fin.read((char*)&a[3], sizeof(float));
//    return std::make_tuple(a[1],a[2],a[3]);
//}

int main()
{
    clock_t tStart = clock();
    Tissue a[X];
    int Nbins;
    float length_voxel;
    //int vertex0[N][3];
    int(*vertex)[3] = nullptr;
    int FLAG;
    int i, j;
    short int* T1=nullptr;
    //vertex = vertex0;


    int threadsPerBlock = M;
    int numBlocks = 1;
    //a[0] = { 0,"air",0.0001,1,1.0 ,1};
    float *k=nullptr;
   // char directory2[] = "C:\\Users\\Administrator\\source\\data\\para.bin";
    std::tie(Nbins, length_voxel,FLAG,T1,k,vertex) = main_gen(T1,k,vertex);
    std::cout << "received\n";
    a[0] = { 0,"air",     0.01,  0.01,   0 ,   1 };
    a[1] = { 1,"midium1", *(k+1), *(k + 2), *(k + 3) ,1.333};
    int* seedofseed = (int*)malloc(sizeof(int) * int(AM / M));

    


    hipError_t c;

    std::random_device rd;  // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    std::uniform_real_distribution<> dis(0.0, 1.0);

    std::string directory0 = "./data/model_slab.dat";
    std::string directory1 = "./data/path_slab.dat";
    std::vector< std::vector<float> >(*path);
    //std::vector< std::vector<float> > path_AM[AM];

    
    /*std::tie(Nbins, length_voxel, T1,
        FLAG, vertex) = read_model(directory0);*/
    
    write_path(directory1);

    /*seedofseed[0] = 1145141919;
    for (i = 0;i < int(AM/M)-1;i++) {
        seedofseed[i + 1] = Rand01(seedofseed[i]);
        std::cout << "dis = " << seedofseed[i+1] << "\n";
    }*/
    /*for (i = 0;i < int(AM / M);i++) {
        seedofseed[i] = dis(gen);
        std::cout << "dis = " << seedofseed[i] << "\n";
    }*/

    for (j = 0;j < (AM / M);j++) {

        auto path2 = new float[M][K][5];
        int* rndseed = (int*)malloc(sizeof(int) * M);

        short int* T1_gpu;
        Tissue* a_gpu;
        int(*vertex_gpu)[3];
        float(*path2_gpu)[K][5];
        float(*c0)[K][5];
        int* rndseed_gpu;
        float(*diss)[N][2];
        float(*face1)[3];
        float(*face2)[3];
        float(*face3)[3];
        float(*d1)[3], (*d2)[3], (*di)[3], (*n1)[3], (*n2)[3], (*vrot)[3], (*temp0)[3], (*temp2)[3];

        hipMalloc((void**)&diss, M * N * 2 * sizeof(float));
        hipMalloc((void**)&face1, M * 3 * sizeof(float));
        hipMalloc((void**)&face2, M * 3 * sizeof(float));
        hipMalloc((void**)&face3, M * 3 * sizeof(float));
        hipMalloc((void**)&d1, M * 3 * sizeof(float));
        hipMalloc((void**)&d2, M * 3 * sizeof(float));
        hipMalloc((void**)&di, M * 3 * sizeof(float));
        hipMalloc((void**)&n1, M * 3 * sizeof(float));
        hipMalloc((void**)&n2, M * 3 * sizeof(float));
        hipMalloc((void**)&vrot, M * 3 * sizeof(float));
        hipMalloc((void**)&temp0, M * 3 * sizeof(float));
        hipMalloc((void**)&temp2, M * 3 * sizeof(float));

        hipMalloc(&T1_gpu, pow(Nbins, 3) * sizeof(short int));
        hipMalloc(&a_gpu, sizeof(a));
        hipMalloc(&vertex_gpu, N * 3 * sizeof(int));
        hipMalloc(&path2_gpu, M * K * 5 * sizeof(float));
        hipMalloc(&c0, M * K * 5 * sizeof(float));
        hipMalloc(&rndseed_gpu, M * sizeof(int));

        /*rndseed[0] = dis(gen)* 2147483647;
        for (i = 0;i < M-1;i++) {
            rndseed[i + 1] = Rand0(rndseed[i]);

        }*/
        for (i = 0;i < M;i++) {
            rndseed[i] = dis(gen) * 2147483647;

        }

        hipMemcpy(T1_gpu, T1, pow(Nbins, 3) * sizeof(short int), hipMemcpyHostToDevice);
        hipMemcpy(a_gpu, a, sizeof(a), hipMemcpyHostToDevice);
        hipMemcpy(vertex_gpu, vertex, N * 3 * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(rndseed_gpu, rndseed, M * sizeof(int), hipMemcpyHostToDevice);

        simulate << <1, M >> > (T1_gpu, Nbins, length_voxel,
            a_gpu, FLAG, vertex_gpu, path2_gpu, c0, rndseed_gpu, diss, face1, face2, face3,
            d1, d2, di, n1, n2, vrot, temp0, temp2);
        //thrust::tie(path2) = simulate(T1, Nbins, length_voxel, a, FLAG, vertex, path0,path2);
        c = hipDeviceSynchronize();
        std::cout << "c : " << c << "\n";

        hipMemcpy(path2, path2_gpu, M * K * 5 * sizeof(float), hipMemcpyDeviceToHost);
        path = array2vector(path2, 0);
        //memcpy(&*(path_AM + j * M), path,sizeof(path)*M);

        write_path2(path, directory1);

        array2vector(path2, 1);

        hipFree(T1_gpu);
        hipFree(a_gpu);
        hipFree(vertex_gpu);
        hipFree(path2_gpu);
        hipFree(c0);
        hipFree(rndseed_gpu);
        hipFree(diss);
        hipFree(face1);hipFree(face2);hipFree(face3);
        hipFree(d1);hipFree(d2);hipFree(di);
        hipFree(n1);hipFree(n2);hipFree(vrot);hipFree(temp0);hipFree(temp2);
        free(path2);
        free(rndseed);
    }
    free(seedofseed);
    
    

    printf("Time taken: %.2fs\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);
    return 0;
}
